#include <hip/hip_runtime.h>
#include <optix.h>
#include <optix_stubs.h>
#include <optix_function_table_definition.h>

#include "gpu/exceptions.hpp"

OptixDeviceContext initializeOptix() {
    OptixDeviceContext context = nullptr;

    // Initialize CUDA with a no-op call to the the CUDA runtime API
    CUDA_CHECK(hipFree(0));

    // Take the current context.
    hipCtx_t cuContext = 0;
    OPTIX_CHECK(optixInit());
    OptixDeviceContextOptions options = {};
    OPTIX_CHECK(optixDeviceContextCreate(cuContext, &options, &context));

    return context;
}