#include <hip/hip_runtime.h>
#include <optix.h>
#include <optix_stubs.h>
#include <optix_function_table_definition.h>

void initialize_optix() {
    OptixDeviceContext context = nullptr;

    // Initialize CUDA with a no-op call to the the CUDA runtime API
    hipFree(0);

    // Take the current context.
    hipCtx_t cuContext = 0;
    optixInit();
    OptixDeviceContextOptions options = {};
    optixDeviceContextCreate(cuContext, &options, &context);
}